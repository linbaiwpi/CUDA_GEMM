
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024 // 3*1024*1024
#define THREAD_PER_BLOCK 256

__global__ void reduce_kernel(float *input, float *output) {
    /*
    // printf("input: %f\n", input[threadIdx.x]);
    float *input_begin = input + blockIdx.x * blockDim.x;
    for (int i = 1; i < blockDim.x; i *= 2) {
        if (threadIdx.x % (i * 2) == 0) {
            input_begin[threadIdx.x] += input_begin[threadIdx.x + i];
            // printf("threadIdx.x : %d = %f\n", threadIdx.x, input_begin[threadIdx.x]);
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        output[blockIdx.x] = input_begin[0];
        // printf("%f\n", output[blockIdx.x]);
    }
    */
    /*
    int tid = threadIdx.x;
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i=1; i<blockDim.x; i++) {
        if (tid % (2 * i) == 0)
            input[idx] += input[idx+i];
        __syncthreads();
    }
    if (tid==0)
        output[blockIdx.x] = input[tid];
    */

    float *input_begin = input + blockDim.x * blockIdx.x;
    for (int i = 1; i < blockDim.x; i *= 2) {
        if (threadIdx.x % (2 * i) == 0)
            input_begin[threadIdx.x] += input_begin[threadIdx.x + i];
        __syncthreads();
    }
    if (threadIdx.x==0)
        output[blockIdx.x] = *input_begin;
}

int main(int argc, char **argv) {
    float *input, *d_input;
    float *output, *d_output;
    float *result;

    // cpu malloc
    input = (float *)malloc(N * sizeof(float));
    int block_num = (N + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;
    output = (float *)malloc(block_num * sizeof(float));
    result = (float *)malloc(block_num * sizeof(float));
    // cuda malloc
    hipMalloc((void **)&d_input, N * sizeof(float));
    hipMalloc((void **)&d_output, block_num * sizeof(float));


    // cpu assigment values
    for (int i = 0; i< N ; ++i) {
        input[i] = 2.0 * (float)drand48() - 1.0;
    }
    for (int i = 0; i< block_num; ++i) {
        float curr = 0;
        for (int j = 0; j < THREAD_PER_BLOCK; ++j) {
            if (i * THREAD_PER_BLOCK + j < N)
                curr += input[i * THREAD_PER_BLOCK + j];
        }
        result[i] = curr;
    }


    // cpoy value into cuda
    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid_dim(block_num);
    dim3 block_dim(THREAD_PER_BLOCK);
    printf("%d\n", block_num);
    printf("%d\n", THREAD_PER_BLOCK);
    reduce_kernel<<<grid_dim, block_dim>>>(d_input, d_output);

    hipMemcpy(output, d_output, block_num*sizeof(float), hipMemcpyDeviceToHost);

    printf("RESULT ===== \n");
    for (int i = 0; i < block_num; ++i) {
        printf("%2f, %2f, %2f\n", output[i], result[i], (output[i] - result[i]));
    }

    free(input);
    free(output);
    free(result);
    hipFree(d_input);
    hipFree(d_output);

    printf("Reduce\n");
    return 0;
}

